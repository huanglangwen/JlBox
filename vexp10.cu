
#include <hip/hip_runtime.h>
//#include <helper_math.h>

extern "C" {

__global__ void kernel_vexp10(double *v)
{
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    v[i]=exp10(v[i]);
}

}
